#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <float.h>
#include "cluster_cuda.cuh"

// This assumes address stores the average of n elements atomically updates
// address to store the average of n + 1 elements (the n elements as well as
// val). This might be useful for updating cluster centers.
// modified from http://stackoverflow.com/a/17401122
__device__ 
float atomicUpdateAverage(float* address, int n, float val)
{
  int* address_as_i = (int*) address;
  int old = *address_as_i;
  int assumed;
  do {
    assumed = old;
    float next_val = (n * __int_as_float(assumed) + val) / (n + 1);
    old = ::atomicCAS(address_as_i, assumed,
		      __float_as_int(next_val));
  } while (assumed != old);
  return __int_as_float(old);
}

// computes the distance squared between vectors a and b where vectors have
// length size and stride stride.
__device__ 
float squared_distance(float *a, float *b, int stride, int size) {
  float dist = 0.0;
  for (int i=0; i < size; i++) {
    float diff = a[stride * i] - b[stride * i];
    dist += diff * diff;
  }
  return dist;
}

/*
 * Notationally, all matrices are column majors, so if I say that matrix Z is
 * of size m * n, then the stride in the m axis is 1. For purposes of
 * optimization (particularly coalesced accesses), you can change the format of
 * any array.
 *
 * clusters is a REVIEW_DIM * k array containing the location of each of the k
 * cluster centers.
 *
 * cluster_counts is a k element array containing how many data points are in 
 * each cluster.
 *
 * k is the number of clusters.
 *
 * data is a REVIEW_DIM * batch_size array containing the batch of reviews to
 * cluster. Note that each review is contiguous (so elements 0 through 49 are
 * review 0, ...)
 *
 * output is a batch_size array that contains the index of the cluster to which
 * each review is the closest to.
 *
 * batch_size is the number of reviews this kernel must handle.
 */
__global__
void sloppyClusterKernel(float *clusters, int *cluster_counts, int k, 
                          float *data, int *output, int batch_size) {
    // TODO: write me
    // Access one element of the batch at a time...
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < batch_size) {
        float *this_review = data[index * REVIEW_DIM];
        
        // Find the closest cluster
        int closest_cluster = 0;
        float smallest_distance = FLT_MAX;
        for (int i = 0; i < k; i++) {
            float *cluster = clusters[i * REVIEW_DIM];
            float distance = squared_distance(this_review, cluster, 1, REVIEW_DIM);
            if (distance < smallest_distance) {
                closest_cluster = i;
                smallest_distance = distance;
            }
        }
        
        // Assign this_review to the closest cluster
        output[index] = closest_cluster;
        
        // update said cluster
        float *cluster = clusters[closest_cluster * REVIEW_DIM];
        int cluster_size = cluster_counts[closest_cluster];
        for (int i = 0; i < REVIEW_DIM; i++) {
            float newAvg = atomicUpdateAverage(cluster[i], cluster_size, this_review[i]);
            cluster[i] = newAvg;
        }
        
        // Update the cluster size
        cluster_counts[closest_cluster] = cluster_size + 1;        
        
        index += blockDim.x * gridDim.x;
    }
}


void cudaCluster(float *clusters, int *cluster_counts, int k,
		 float *data, int *output, int batch_size, 
		 hipStream_t stream) {
  int block_size = (batch_size < 1024) ? batch_size : 1024;

  // grid_size = CEIL(batch_size / block_size)
  int grid_size = (batch_size + block_size - 1) / block_size;
  int shmem_bytes = 0;

  sloppyClusterKernel<<<
    block_size, 
    grid_size, 
    shmem_bytes, 
    stream>>>(clusters, cluster_counts, k, data, output, batch_size);
}
