#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "Blur_cuda.cuh"


__global__
void
cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    int N, int blur_v_size) {

    /* TODO: Fill in the implementation for the GPU-accelerated convolution. 

    It may be helpful to use the information in the lecture slides, 
    as well as the CPU implementation, as a reference. */
    /*{
        for (int i = 0; i < blur_v_size; i++){
            for (int j = 0; j <= i; j++){
                out_data[i] += raw_data[i - j] * blur_v[j]; 
            }
        }
        for (int i = blur_v_size; i < N; i++){
            for (int j = 0; j < blur_v_size; j++){
                out_data[i] += raw_data[i - j] * blur_v[j]; 
            }
        }
    }*/
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < N) {
        for (int j = 0; j <= index && j < blur_v_size; j++){
            out_data[index] += raw_data[index - j] * blur_v[j];
        }
        index += blockDim.x * gridDim.x;
    }
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int N,
        const unsigned int blur_v_size) {
        
    /* TODO: Call the kernel above this function. */
    cudaBlurKernel<<<blocks, threadsPerBlock>>>
        (raw_data, blur_v, out_data, N, blur_v_size);
}
