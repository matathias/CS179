#include "hip/hip_runtime.h"
/* CUDA finite difference wave equation solver, written by
 * Jeff Amelang, 2012
 *
 * Modified by Kevin Yuh, 2013-14 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "Cuda1DFDWave_cuda.cuh"


/* TODO: You'll need a kernel here, as well as any helper functions
to call it */

__global__
void waveEquationKernal(float *old_data, float *current_data, float *new_data,
                        int numberOfNodes, float constant) {
    
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < numberOfNodes - 1) {
        // This is to make sure that thread index 0 can still move on to the
        // next thread at blockDim.x * gridDim.x
        if (index > 0) {
            // Wave Equation!
            // y_x,t+1 = 2*y_x,t - y_x,t-1 + 
            //                       (c*dt/dx)^2 * (y_x+1,t - 2*y_x,t + y_x-1,t)
            new_data[index] = 2 * current_data[index] 
                              - old_data[index]
                              + constant 
                              * (current_data[index + 1] 
                                 - 2 * current_data[index] 
                                 + current_data[index - 1]);
        }
        index += blockDim.x * gridDim.x;
    }
}

void waveEquation(float *old_data, float *current_data, float *new_data,
                  int numberOfNodes, float c, float dt, float dx,
                  int blocks, int threadsPerBlock) {

    float constant = ((c * c * dt * dt) / (dx * dx));
    waveEquationKernal<<<blocks, threadsPerBlock>>>(old_data, current_data,
                                                    new_data, numberOfNodes,
                                                    constant);
}
